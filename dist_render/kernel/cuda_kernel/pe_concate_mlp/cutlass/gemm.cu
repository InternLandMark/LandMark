#include <torch/extension.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <iostream>
#include <stdlib.h>
#include <vector>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/embed.h>

#include <iostream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"

#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/gemm.h"
#include "helper.h"


////////////////////////////////////////////////////////////////////////////////


void gemm(torch::Tensor Ga, torch::Tensor Gb0,  torch::Tensor Gd0) {


  cutlass::gemm::GemmCoord problem_size_0(Ga.sizes()[0], Gb0.sizes()[0], Ga.sizes()[1]);
  std::cout << "problem_size_0: " << problem_size_0.mnk() << "\n";
  //
  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = cutlass::half_t;
  using ElementCompute = cutlass::half_t;

  ElementCompute alpha0 = ElementCompute(1);
  ElementCompute beta0 = ElementCompute(0); //beta=1 for bias
  // attention: 这里的第二项"128"需要与problem_size_0.n()对齐, 但不可使用变量, 必须手动设置
  using ThreadblockShape0 = cutlass::gemm::GemmShape<128, 128, 32>;
  using WarpShape0 = cutlass::gemm::GemmShape<64, 64, 32>;
  // attention: 这里的第二项"128"需要与gemm_f16_sm80_problem_size_1.n()对齐, 但不可使用变量, 必须手动设置
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  //D = α * A * B + β * C  其中，标量α = β = 1
  using Gemm0 = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor, // A
    cutlass::half_t,
    cutlass::layout::ColumnMajor, // B
    ElementOutput,
    cutlass::layout::RowMajor,    // C
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    ThreadblockShape0,
    WarpShape0,
    InstructionShape,
    cutlass::epilogue::thread::LinearCombinationRelu<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementCompute,
      cutlass::epilogue::thread::ScaleType::NoBetaScaling
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<1>,
    3
  >;


  cutlass::half_t* d_at = (cutlass::half_t*)Ga.data_ptr();
  cutlass::half_t* d_b0t = (cutlass::half_t*)Gb0.data_ptr();
  cutlass::half_t* d_d0t = (cutlass::half_t*)Gd0.data_ptr();


  typename Gemm0::Arguments arguments_0(
      problem_size_0,
      {d_at, problem_size_0.k()},
      // tensor_B0.device_ref(),
      {d_b0t, problem_size_0.k()},
      {},
      // tensor_D0.device_ref(),
      {d_d0t, problem_size_0.n()},
      {alpha0, beta0}
    );
    Gemm0 gemm_op_0;
    // hipEvent_t start, stop1, stop2;
    //     hipEventCreate(&start);
    //     hipEventCreate(&stop1);

    //     hipEventRecord(start);

    cutlass::Status status = gemm_op_0.initialize(arguments_0);
    CUTLASS_CHECK(status);
    // Run the GEMM
    status = gemm_op_0();
    CUTLASS_CHECK(status);

    // hipEventRecord(stop1);
    // hipDeviceSynchronize();
    // float gemm0Time;
    // hipEventElapsedTime(&gemm0Time, start, stop1);
    // std::cout << "gemm  time " << gemm0Time << " ms\n";



  return;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    m.def("run", &gemm, "gemm based on cutlass");
    //"Gpu_Cublas"代表python中对应的函数，&np_multiply_Cublas是对应的C++函数指针，之后的字符串是python中的函数doc
}



////////////////////////////////////////////////////////////////////////////////
